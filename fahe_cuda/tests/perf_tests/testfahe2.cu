#include "hip/hip_runtime.h"
#include <criterion/criterion.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#include "fahe2.h"
#include "helper.h"
#include "logger.h"

// TestSuite(fahe2, .init = thread_setup, .fini = thread_teardown);

Test(fahe2, fahe2_analysis_fahe2full) {
  int num_trials = 1000;
  fahe_params params = {128, 32, 10, 32};
  int list_size = 32;
  BIGNUM *bn_list_size = BN_new();
  BN_CTX *ctx = BN_CTX_new();

  if (!bn_list_size || !ctx) {
    log_message(LOG_FATAL, "Memory allocation failed\n");
    exit(EXIT_FAILURE);
  }

  BN_set_word(bn_list_size, list_size);

  const char *filename = "../assets/mintest.txt";
  BIGNUM **msg_list = read_bignum_list_from_file(filename, &list_size);
  if (!msg_list) {
    log_message(LOG_FATAL, "Failed to read BIGNUM list from file\n");
    BN_free(bn_list_size);
    BN_CTX_free(ctx);
    exit(EXIT_FAILURE);
  }

  double total_keygen_time = 0.0;
  double total_encryption_time = 0.0;
  double total_decryption_time = 0.0;

  for (int trial = 0; trial < num_trials; trial++) {
    // TIMED KEYGEN
    clock_t fahe2_keygen_start_time = clock();
    fahe2 *fahe2_instance = fahe2_init(&params);
    clock_t fahe2_keygen_end_time = clock();

    if (!fahe2_instance) {
      log_message(LOG_FATAL, "Failed to initialize FAHE2 instance\n");
      break;
    }

    double fahe2_keygen_time =
        (double)(fahe2_keygen_end_time - fahe2_keygen_start_time) /
        CLOCKS_PER_SEC;
    total_keygen_time += fahe2_keygen_time;

    // TIMED ENCRYPTION
    clock_t fahe2_encryption_start_time = clock();
    BIGNUM **ciphertext_list =
        fahe2_encrypt_list(fahe2_instance->key, msg_list,
                           BN_get_word(fahe2_instance->num_additions), ctx);
    clock_t fahe2_encryption_end_time = clock();

    if (!ciphertext_list) {
      log_message(LOG_FATAL, "Encryption failed\n");
      fahe2_free(fahe2_instance);
      break;
    }

    double fahe2_encryption_time =
        (double)(fahe2_encryption_end_time - fahe2_encryption_start_time) /
        CLOCKS_PER_SEC;
    total_encryption_time += fahe2_encryption_time;

    // TIMED DECRYPTION
    clock_t fahe2_decryption_start_time = clock();
    BIGNUM **decrypted_msg_list = fahe2_decrypt_list(
        fahe2_instance->key, ciphertext_list, bn_list_size, ctx);
    clock_t fahe2_decryption_end_time = clock();

    if (!decrypted_msg_list) {
      log_message(LOG_FATAL, "Decryption failed\n");
      for (unsigned int i = 0; i < list_size; i++) {
        BN_free(ciphertext_list[i]);
      }
      free(ciphertext_list);
      fahe2_free(fahe2_instance);
      break;
    }

    double fahe2_decryption_time =
        (double)(fahe2_decryption_end_time - fahe2_decryption_start_time) /
        CLOCKS_PER_SEC;
    total_decryption_time += fahe2_decryption_time;

    // Free resources for this trial
    for (unsigned int i = 0; i < list_size; i++) {
      BN_free(ciphertext_list[i]);
      BN_free(decrypted_msg_list[i]);
    }
    free(ciphertext_list);
    free(decrypted_msg_list);
    fahe2_free(fahe2_instance);
  }

  // Calculate averages
  double avg_keygen_time = total_keygen_time / num_trials;
  double avg_encryption_time = total_encryption_time / num_trials;
  double avg_decryption_time = total_decryption_time / num_trials;

  print_test_table("FAHE2 Test", params, num_trials, avg_keygen_time,
                   avg_encryption_time, avg_decryption_time, total_keygen_time,
                   total_encryption_time, total_decryption_time);

  // Free the last instance of bn_list_size and msg_list
  BN_free(bn_list_size);
  for (unsigned int i = 0; i < list_size; i++) {
    BN_free(msg_list[i]);
  }
  free(msg_list);
  BN_CTX_free(ctx);
}