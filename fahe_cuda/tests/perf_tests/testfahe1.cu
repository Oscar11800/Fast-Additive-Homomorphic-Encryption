#include "hip/hip_runtime.h"
#include <criterion/criterion.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#include "fahe1.cuh"
#include "helper.cuh"
#include "logger.h"

Test(fahe1, fahe1_analysis_fahe1_full) {
  // Number of trials
  int num_trials = 100;
  // lambda, m_max, alpha, msg_size
  fahe_params params = {128, 32, 6, 32};
  int list_size = 32;
  BIGNUM *bn_list_size = BN_new();
  BN_set_word(bn_list_size, list_size);

  const char *filename = "../assets/mintest.txt";
  BIGNUM **msg_list = read_bignum_list_from_file(filename, &list_size);

  double total_keygen_time = 0.0;
  double total_encryption_time = 0.0;
  double total_decryption_time = 0.0;

  for (int trial = 0; trial < num_trials; trial++) {
    // TIMED KEYGEN
    clock_t fahe1_keygen_start_time = clock();
    fahe1 *fahe1_instance = fahe1_init(&params);
    clock_t fahe1_keygen_end_time = clock();

    double fahe1_keygen_time =
        (double)(fahe1_keygen_end_time - fahe1_keygen_start_time) /
        CLOCKS_PER_SEC;
    total_keygen_time += fahe1_keygen_time;

    // TIMED ENCRYPTION
    clock_t fahe1_encryption_start_time = clock();
    BIGNUM **ciphertext_list = fahe1_encrypt_list(
        fahe1_instance->key.p, fahe1_instance->key.X, fahe1_instance->key.rho,
        fahe1_instance->key.alpha, msg_list, fahe1_instance->num_additions);
    clock_t fahe1_encryption_end_time = clock();
    double fahe1_encryption_time =
        (double)(fahe1_encryption_end_time - fahe1_encryption_start_time) /
        CLOCKS_PER_SEC;
    total_encryption_time += fahe1_encryption_time;

    // TIMED DECRYPTION
    clock_t fahe1_decryption_start_time = clock();
    BIGNUM **decrypted_msg_list =
        fahe1_decrypt_list(fahe1_instance->key.p, fahe1_instance->key.m_max,
                           fahe1_instance->key.rho, fahe1_instance->key.alpha,
                           ciphertext_list, bn_list_size);
    clock_t fahe1_decryption_end_time = clock();
    double fahe1_decryption_time =
        (double)(fahe1_decryption_end_time - fahe1_decryption_start_time) /
        CLOCKS_PER_SEC;
    total_decryption_time += fahe1_decryption_time;

    // Free resources for this trial
    for (unsigned int i = 0; i < list_size; i++) {
      BN_free(ciphertext_list[i]);
      BN_free(decrypted_msg_list[i]);
    }
    free(ciphertext_list);
    free(decrypted_msg_list);
    fahe1_free(fahe1_instance);
  }

  // Calculate averages
  double avg_keygen_time = total_keygen_time / num_trials;
  double avg_encryption_time = total_encryption_time / num_trials;
  double avg_decryption_time = total_decryption_time / num_trials;
  print_test_table("FAHE1 Test", params, num_trials, avg_keygen_time,
                   avg_encryption_time, avg_decryption_time, total_keygen_time,
                   total_encryption_time, total_decryption_time);

  // Free the last instance of bn_list_size and msg_list
  BN_free(bn_list_size);
  for (unsigned int i = 0; i < list_size; i++) {
    BN_free(msg_list[i]);
  }
  free(msg_list);
}
